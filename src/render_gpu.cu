#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>

#include "render_gpu.hpp"

struct rgba8_t {
  std::uint8_t r;
  std::uint8_t g;
  std::uint8_t b;
  std::uint8_t a;
};

rgba8_t heat_lut(float x)
{
  assert(0 <= x && x <= 1);
  float x0 = 1.f / 4.f;
  float x1 = 2.f / 4.f;
  float x2 = 3.f / 4.f;

  if (x < x0)
  {
    auto g = static_cast<std::uint8_t>(x / x0 * 255);
    return rgba8_t{0, g, 255, 255};
  }
  else if (x < x1)
  {
    auto b = static_cast<std::uint8_t>((x1 - x) / x0 * 255);
    return rgba8_t{0, 255, b, 255};
  }
  else if (x < x2)
  {
    auto r = static_cast<std::uint8_t>((x - x1) / x0 * 255);
    return rgba8_t{r, 255, 0, 255};
  }
  else
  {
    auto b = static_cast<std::uint8_t>((1.f - x) / x0 * 255);
    return rgba8_t{255, b, 0, 255};
  }
}


__global__
void mandel_iter(int *iter_matrix, int width, int height, int n_iterations)
{
    const unsigned int offset = blockIdx.x*blockDim.x + threadIdx.x;
    int X = offset % width;
    int Y = (offset-X)/width;

    int idx = width * Y + X;

    float x0 = ((float)X / width) * (3.5) - 2.5;
    float y0 = ((float)Y /height) * (2) - 1;

    float x = 0.0f;
    float y = 0.0f;

    int iter = 0;
    float xtemp;
    while((x *x + y *y <= 4.0f) && (iter < n_iterations)){
      xtemp = x * x - y * y + x0;
      y = 2.0f * x * y + y0;
      x = xtemp;
      iter++;
    }

    iter_matrix[idx] = iter;
}


void GPURenderer::render_gpu(std::byte* buffer,
                             int width,
                             int height,
                             std::ptrdiff_t stride,
                             int n_iterations)
{
    //int *histogram_cu;
    int *histogram = new int[n_iterations];
    for (int i = 0; i < n_iterations; ++i)
        histogram[i] = 0;

    int N = width * height;
    int *iter_matrix_cu;
    int *iter_matrix = new int[N];

    float total = 0.f;

    hipMalloc(&iter_matrix_cu, N*sizeof(int));

   dim3 nb_blocks(ceil(float(height)/32),(float(width)/1),1);
   dim3 threads_per_block(32, 1, 1);

    mandel_iter<<<nb_blocks, threads_per_block>>>(iter_matrix_cu,
                                                  //histogram_cu,
                                                  width, height,
                                                  n_iterations);

    hipMemcpy(iter_matrix_cu, iter_matrix, N*sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++)
    {
        if (iter_matrix[i] != 0)
            std::cout << iter_matrix[i] << std::endl;
    }

    hipFree(iter_matrix_cu);

    rgba8_t *hue = new rgba8_t[n_iterations + 1];
    for (int i = 0; i < n_iterations + 1; ++i)
        hue[i] = rgba8_t{0, 0, 0, 255};
    float tmp = (float)histogram[0] / total;
    hue[0] = heat_lut(tmp);
    for (int i = 1; i < n_iterations; ++i)
    {
        tmp = tmp + ((float)histogram[i] / total);
        hue[i] = heat_lut(tmp);
    }
 
    auto buffer_down = buffer + stride * (height - 1);
    for (int Py = 0; Py < height / 2; ++Py)
    {
        rgba8_t* lineptr_top = reinterpret_cast<rgba8_t*>(buffer);
        rgba8_t* lineptr_bottom = reinterpret_cast<rgba8_t*>(buffer_down);
        for (int Px = 0; Px < width; ++Px)
        {
            lineptr_top[Px] = hue[iter_matrix[Py * width + Px]];
            lineptr_bottom[Px] = hue[iter_matrix[Py * width + Px]];
        }
        buffer += stride;
        buffer_down -= stride;
    }

    delete[] iter_matrix;
}
